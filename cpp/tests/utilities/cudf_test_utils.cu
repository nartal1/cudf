/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Copyright 2019 BlazingDB, Inc.
 *     Copyright 2019 Eyal Rozenberg <eyalroz@blazingdb.com>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cudf_test_utils.cuh"
#include <nvstrings/NVCategory.h>
#include <nvstrings/NVStrings.h>
#include <utilities/type_dispatcher.hpp>

namespace {

namespace detail {

// When streaming char-like types, the standard library streams tend to treat
// them as characters rather than numbers, e.g. you would get an 'a' instead of 97.
// The following function(s) ensure we "promote" such values to integers before
// they're streamed

template <typename T>
const T& promote_for_streaming(const T& x) { return x; }


//int promote_for_streaming(const char& x)          { return x; }
//int promote_for_streaming(const unsigned char& x) { return x; }
int promote_for_streaming(const signed char& x)   { return x; }

} // namespace detail


struct column_printer {
    template<typename Element>
    void operator()(gdf_column const* the_column, unsigned min_printing_width)
    {
        gdf_size_type num_rows { the_column->size };

        Element const* column_data { static_cast<Element const*>(the_column->data) };

        std::vector<Element> host_side_data(num_rows);
        hipMemcpy(host_side_data.data(), column_data, num_rows * sizeof(Element), hipMemcpyDeviceToHost);

        gdf_size_type const num_masks { gdf_valid_allocation_size(num_rows) };
        std::vector<gdf_valid_type> h_mask(num_masks, ~gdf_valid_type { 0 });
        if (nullptr != the_column->valid) {
            hipMemcpy(h_mask.data(), the_column->valid, num_masks * sizeof(gdf_valid_type), hipMemcpyDeviceToHost);
        }

        for (gdf_size_type i = 0; i < num_rows; ++i) {
            std::cout << std::setw(min_printing_width);
            if (gdf_is_valid(h_mask.data(), i)) {
                std::cout << detail::promote_for_streaming(host_side_data[i]);
            }
            else {
                std::cout << null_representative;
            }
            std::cout << ' ';
        }
        std::cout << std::endl;

        if(the_column->dtype == GDF_STRING_CATEGORY){
            std::cout<<"Data on category:\n";
            size_t length = 1;

            if(the_column->dtype_info.category != nullptr){
                size_t keys_size = static_cast<NVCategory *>(the_column->dtype_info.category)->keys_size();
                if(keys_size>0){
                    char ** data = new char *[keys_size];
                    for(size_t i=0; i<keys_size; i++){
                        data[i]=new char[length+1];
                    }

                    static_cast<NVCategory *>(the_column->dtype_info.category)->get_keys()->to_host(data, 0, keys_size);

                    for(size_t i=0; i<keys_size; i++){
                        data[i][length]=0;
                    }

                    for(size_t i=0; i<keys_size; i++){
                        std::cout<<"("<<data[i]<<"|"<<i<<")\t";
                    }
                    std::cout<<std::endl;
                }
            }
        }
    }
};

/**---------------------------------------------------------------------------*
 * @brief Functor for comparing if two elements between two gdf_columns are
 * equal.
 *
 *---------------------------------------------------------------------------**/
template <typename T, bool has_nulls>
struct elements_equal {
  gdf_column lhs_col;
  gdf_column rhs_col;
  bool nulls_are_equivalent;

  using bit_mask_t = bit_mask::bit_mask_t;

  /**---------------------------------------------------------------------------*
   * @brief Constructs functor for comparing elements between two gdf_column's
   *
   * @param lhs The left column for comparison
   * @param rhs The right column for comparison
   * @param nulls_are_equal Desired behavior for whether or not nulls are
   * treated as equal to other nulls. Defaults to true.
   *---------------------------------------------------------------------------**/
  __host__ __device__ elements_equal(gdf_column lhs, gdf_column rhs,
                                     bool nulls_are_equal = true)
      : lhs_col{lhs}, rhs_col{rhs}, nulls_are_equivalent{nulls_are_equal} {}

  __device__ bool operator()(gdf_index_type row) {    
    bool const lhs_is_valid{gdf_is_valid(lhs_col.valid, row)};
    bool const rhs_is_valid{gdf_is_valid(rhs_col.valid, row)};

    if (lhs_is_valid and rhs_is_valid) {
      return static_cast<T const*>(lhs_col.data)[row] ==
             static_cast<T const*>(rhs_col.data)[row];
    }

    // If one value is valid but the other is not
    if (lhs_is_valid != rhs_is_valid) {
      return false;
    }

    return nulls_are_equivalent;
  }
};

} // namespace anonymous

/**
 * ---------------------------------------------------------------------------*
 * @brief Compare two gdf_columns on all fields, including pairwise comparison
 * of data and valid arrays
 *
 * @tparam T The type of columns to compare
 * @param left The left column
 * @param right The right column
 * @return bool Whether or not the columns are equal
 * ---------------------------------------------------------------------------**/
template <typename T>
bool gdf_equal_columns(gdf_column const& left, gdf_column const& right)
{
  if (left.size != right.size) return false;
  if (left.dtype != right.dtype) return false;
  if (left.null_count != right.null_count) return false;
  if (left.dtype_info.time_unit != right.dtype_info.time_unit) return false;

  if ((left.col_name == nullptr) != (right.col_name == nullptr))
    return false; // if one is null but not both

  if (left.col_name != nullptr && std::strcmp(left.col_name, right.col_name) != 0)
    return false;

  if ((left.data == nullptr) != (right.data == nullptr))
    return false;  // if one is null but not both
  
  if ((left.valid == nullptr) != (right.valid == nullptr))
    return false;  // if one is null but not both

  if (left.data == nullptr)
      return true;  // logically, both are null

  // both are non-null...
  bool const has_nulls {(left.valid != nullptr) && (left.null_count > 0)};

  bool equal_data = (has_nulls) ?
    thrust::all_of(rmm::exec_policy()->on(0),
                   thrust::make_counting_iterator(0),
                   thrust::make_counting_iterator(left.size),
                   elements_equal<T, true>{left, right}) :
    thrust::all_of(rmm::exec_policy()->on(0),
                   thrust::make_counting_iterator(0),
                   thrust::make_counting_iterator(left.size),
                   elements_equal<T, false>{left, right});
  
  CHECK_STREAM(0);

  return equal_data;
}

namespace {

struct columns_equal
{
  template <typename T>
  bool operator()(gdf_column const& left, gdf_column const& right) {
    return gdf_equal_columns<T>(left, right);
  }
};

}; // namespace anonymous

// Type-erased version of gdf_equal_columns
bool gdf_equal_columns(gdf_column const& left, gdf_column const& right)
{
  return cudf::type_dispatcher(left.dtype, columns_equal{}, left, right);
}

void print_gdf_column(gdf_column const * the_column, unsigned min_printing_width)
{
  cudf::type_dispatcher(the_column->dtype, column_printer{}, 
                        the_column, min_printing_width);
}

void print_valid_data(const gdf_valid_type *validity_mask,
                      const size_t num_rows)
{
  hipError_t error;
  hipPointerAttribute_t attrib;
  hipPointerGetAttributes(&attrib, validity_mask);
  error = hipGetLastError();

  std::vector<gdf_valid_type> h_mask(gdf_valid_allocation_size(num_rows));
  if (error != hipErrorInvalidValue && isDeviceType(attrib))
    hipMemcpy(h_mask.data(), validity_mask, gdf_valid_allocation_size(num_rows),
               hipMemcpyDeviceToHost);
  else
    memcpy(h_mask.data(), validity_mask, gdf_valid_allocation_size(num_rows));

  std::transform(
      h_mask.begin(), h_mask.begin() + gdf_num_bitmask_elements(num_rows),
      std::ostream_iterator<std::string>(std::cout, " "), [](gdf_valid_type x) {
        auto bits = std::bitset<GDF_VALID_BITSIZE>(x).to_string('@');
        return std::string(bits.rbegin(), bits.rend());
      });
  std::cout << std::endl;
}

gdf_size_type count_valid_bits_host(
    std::vector<gdf_valid_type> const& masks, gdf_size_type const num_rows)
{
  if ((0 == num_rows) || (0 == masks.size())) {
    return 0;
  }

  gdf_size_type count{0};

  // Count the valid bits for all masks except the last one
  for (gdf_size_type i = 0; i < (gdf_num_bitmask_elements(num_rows) - 1); ++i) {
    gdf_valid_type current_mask = masks[i];

    while (current_mask > 0) {
      current_mask &= (current_mask - 1);
      count++;
    }
  }

  // Only count the bits in the last mask that correspond to rows
  int num_rows_last_mask = num_rows % GDF_VALID_BITSIZE;
  if (num_rows_last_mask == 0) {
    num_rows_last_mask = GDF_VALID_BITSIZE;
  }

  // Mask off only the bits that correspond to rows
  gdf_valid_type const rows_mask = ( gdf_valid_type{1} << num_rows_last_mask ) - 1;
  gdf_valid_type last_mask = masks[gdf_num_bitmask_elements(num_rows) - 1] & rows_mask;

  while (last_mask > 0) {
    last_mask &= (last_mask - 1);
    count++;
  }

  return count;
}
