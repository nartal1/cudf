/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cudf/io_readers.hpp"

#include "io/json/json_reader_impl.hpp"

namespace cudf {

JsonReader::JsonReader() noexcept = default;

JsonReader::JsonReader(JsonReader const &rhs) : impl_(std::make_unique<JsonReader::Impl>(rhs.impl_->getArgs())) {}

JsonReader &JsonReader::operator=(JsonReader const &rhs) {
  impl_ = std::make_unique<JsonReader::Impl>(rhs.impl_->getArgs());
  return *this;
}

JsonReader::JsonReader(JsonReader &&rhs) : impl_(std::move(rhs.impl_)) {}

JsonReader &JsonReader::operator=(JsonReader &&rhs) {
  impl_ = std::move(rhs.impl_);
  return *this;
}

JsonReader::JsonReader(json_reader_args const &args) : impl_(std::make_unique<Impl>(args)) {}

table JsonReader::read() {
  if (impl_) {
    return impl_->read();
  } else {
    return table();
  }
}

table JsonReader::read_byte_range(size_t offset, size_t size) {
  if (impl_) {
    return impl_->read_byte_range(offset, size);
  } else {
    return table();
  }
}

JsonReader::~JsonReader() = default;

} // namespace cudf
